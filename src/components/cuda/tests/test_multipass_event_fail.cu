/**
 * @file    test_multipass_event_fail.cu
 * @author  Anustuv Pal
 *          anustuv@icl.utk.edu
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef PAPI
#include "papi.h"
#include "papi_test.h"

#define PASS 1
#define FAIL 0
#define MAX_EVENT_COUNT (32)
#define PRINT(quiet, format, args...) {if (!quiet) {fprintf(stderr, format, ## args);}}
int quiet;

int test_PAPI_add_named_event(int *EventSet, int numEvents, char **EventName) {
    int i, papi_errno;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);
    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_add_named_event(*EventSet, EventName[i]);
        if (papi_errno != PAPI_EMULPASS && papi_errno != PAPI_OK) {
            fprintf(stderr, "Failed to add named event %s with error code %d.\n", EventName[i], papi_errno);
            return FAIL;
        }
    }
    if (papi_errno == PAPI_EMULPASS || papi_errno == PAPI_OK) {
        PRINT(quiet, "PASSED test_PAPI_add_named_event\n");
        return PASS; // Test pass condition
    }
    return FAIL;
}

int test_PAPI_add_event(int *EventSet, int numEvents, char **EventName, int *numEventsSuccessfullyAdded) {
    int event, i, papi_errno;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);

    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_event_name_to_code(EventName[i], &event);
        if (papi_errno != PAPI_OK) {
            fprintf(stderr, "Failed to convert event name %s to event code with error code %d.\n", EventName[i], papi_errno);
            goto fail;
        }
        papi_errno = PAPI_add_event(*EventSet, event);
        if (papi_errno != PAPI_OK) {
            if (papi_errno != PAPI_EMULPASS) {
                fprintf(stderr, "Failed to add event %s with error code %d.\n", EventName[i], papi_errno);
                goto fail;
            }
        }
        else {
            (*numEventsSuccessfullyAdded)++;
        }
    }
    if (papi_errno == PAPI_EMULPASS || papi_errno == PAPI_OK) {
        PRINT(quiet, "PASSED test_PAPI_add_event\n");
        return PASS;
    }
fail:
    return FAIL;
}

int test_PAPI_add_events(int *EventSet, int numEvents, char **EventName, int numEventsSuccessfullyAdded) {
    int papi_errno, i;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);

    int events[MAX_EVENT_COUNT];

    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_event_name_to_code(EventName[i], &events[i]);
        if (papi_errno != PAPI_OK) {
            fprintf(stderr, "Failed to convert event name %s to event code with error code %d.\n", EventName[i], papi_errno);
            goto fail;
        }
    }
    papi_errno = PAPI_add_events(*EventSet, events, numEvents);
    if (papi_errno == PAPI_EMULPASS || papi_errno == PAPI_OK || papi_errno == numEventsSuccessfullyAdded) {
        PRINT(quiet, "PASSED test_PAPI_add_events with %d of %d events succesfully added.\n", numEventsSuccessfullyAdded, numEvents);
        return PASS;
    }

fail:
    return FAIL;
}
#endif

int main(int argc, char **argv)
{
#ifdef PAPI
    int papi_errno, pass;
    int event_set;

    quiet = 0;
    char *test_quiet = getenv("PAPI_CUDA_TEST_QUIET");
    if (test_quiet)
        quiet = (int) strtol(test_quiet, (char**) NULL, 10);

    int event_count = argc - 1;

    /* if no events passed at command line, just report test skipped. */
    if (event_count == 0) {
        fprintf(stderr, "No eventnames specified at command line.\n");
        test_skip(__FILE__, __LINE__, "", 0);
    }

    papi_errno = PAPI_library_init( PAPI_VER_CURRENT );
    if (papi_errno != PAPI_VER_CURRENT) {
        test_fail(__FILE__, __LINE__, "PAPI_library_init() failed", 0);
    }

    papi_errno = PAPI_get_component_index("cuda");
    if (papi_errno < 0 ) {
        test_fail(__FILE__, __LINE__, "CUDA component not configured", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    // Keep track of the number of events from the command line we can actually add
    // This is done to properly check the test in the function test_PAPI_add_events
    int numEventsSuccessfullyAdded = 0;
    pass = test_PAPI_add_event(&event_set, argc-1, argv+1, &numEventsSuccessfullyAdded);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    pass += test_PAPI_add_named_event(&event_set, argc-1, argv+1);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    pass += test_PAPI_add_events(&event_set, argc-1, argv+1, numEventsSuccessfullyAdded);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    if (pass != 3)
        test_fail(__FILE__, __LINE__, "CUDA framework multipass event test failed.", 0);
    else
        test_pass(__FILE__);

    PAPI_shutdown();
#else
    fprintf(stderr, "Please compile with -DPAPI to test this feature.\n");
#endif
    return 0;
}
